#include "hip/hip_runtime.h"
#include <float.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <time.h>

#include "ray.h"
#include "camera.h"
#include "scene.h"
#include "rnd.h"
#include "options.h"

#define STBI_MSC_SECURE_CRT
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

using namespace std;

#include "cudautils.h"
#include "metrics.h"

#include "glwindow.h"

#define DYNAMIC_FETCH_THRESHOLD 20          // If fewer than this active, fetch new rays

const int MaxBlockWidth = 32;
const int MaxBlockHeight = 2; // block width is 32

typedef unsigned long long ull;

__device__ __constant__ float d_colormap[256 * 3];
__device__ __constant__ bvh_node d_nodes[2048];

texture<float4> t_bvh;
texture<float> t_spheres;
float* d_bvh_buf;
float* d_spheres_buf;
vec3* d_fb;
int* d_colors;
unsigned int* d_cuda_render_buffer;

// Camera controls
camera* cam = NULL;
float c_theta = 80 * kPI / 180;
float c_phi = 45 * kPI / 180;
float c_relative_dist = 1.0f;
const float c_rotation_speed = 1 * kPI / 180;
const float c_zoom_speed = 1.0f / 100;
bool camera_updated = false;

struct render_params {
    vec3* fb;
    int leaf_offset;
    unsigned int width;
    unsigned int height;
    unsigned int spp;
    unsigned int maxActivePaths;

    int* colors;

    int maxBounces;
    float lightRadius;
    vec3 lightColor;

    vec3 skyColor;
};

typedef enum pathstate {
    DONE,           // nothing more to do for this path
    SCATTER,        // path need to traverse the BVH tree
    NO_HIT,         // path didn't hit any primitive
    HIT,            // path hit a primitive
    SHADOW,         // path hit a primitive and generated a shadow ray
    HIT_AND_LIGHT  // path hit a primitive and its shadow ray didn't hit any primitive
} pathstate;

struct paths {
    ull* next_sample; // used by init() to track next sample to fetch
    ull* numsamples_perpixel; // how many samples have been traced per pixel so far

    // pixel_id of active paths currently being traced by the renderer, it's a subset of all_sample_pool
    unsigned int* active_paths;
    unsigned int* next_path; // used by hit_bvh() to track next path to fetch and trace

    ray* r;
    ray* shadow;
    rand_state* state;
    vec3* attentuation;
    vec3* emitted;
    unsigned short* bounce;
    pathstate* pstate;
    int* hit_id;
    vec3* hit_normal;
    float* hit_t;

    metrics m;
};

// Renderer
unsigned int r_iteration = 0;
unsigned int r_num_pixels = 0;
paths* r_paths = NULL;

void setup_paths(paths& p, int nx, int ny, int ns, unsigned int maxActivePaths) {
    // at any given moment only kMaxActivePaths at most are active at the same time
    const unsigned num_paths = maxActivePaths;
    checkCudaErrors(hipMalloc((void**)& p.r, num_paths * sizeof(ray)));
    checkCudaErrors(hipMalloc((void**)& p.shadow, num_paths * sizeof(ray)));
    checkCudaErrors(hipMalloc((void**)& p.state, num_paths * sizeof(rand_state)));
    checkCudaErrors(hipMalloc((void**)& p.attentuation, num_paths * sizeof(vec3)));
    checkCudaErrors(hipMalloc((void**)& p.emitted, num_paths * sizeof(vec3)));
    checkCudaErrors(hipMalloc((void**)& p.bounce, num_paths * sizeof(unsigned short)));
    checkCudaErrors(hipMalloc((void**)& p.pstate, num_paths * sizeof(pathstate)));
    checkCudaErrors(hipMalloc((void**)& p.hit_id, num_paths * sizeof(int)));
    checkCudaErrors(hipMalloc((void**)& p.hit_normal, num_paths * sizeof(vec3)));
    checkCudaErrors(hipMalloc((void**)& p.hit_t, num_paths * sizeof(float)));

    checkCudaErrors(hipMalloc((void**)& p.active_paths, num_paths * sizeof(unsigned int)));
    checkCudaErrors(hipMalloc((void**)& p.next_path, sizeof(unsigned int)));

    checkCudaErrors(hipMalloc((void**)& p.next_sample, sizeof(ull)));
    checkCudaErrors(hipMemset((void*)p.next_sample, 0, sizeof(ull)));
    p.m.allocateDeviceMem();

    checkCudaErrors(hipMalloc((void**)&p.numsamples_perpixel, nx * ny * sizeof(ull)));
    checkCudaErrors(hipMemset((void*)p.numsamples_perpixel, 0, nx * ny * sizeof(ull)));
}

void free_paths(const paths& p) {
    checkCudaErrors(hipFree(p.r));
    checkCudaErrors(hipFree(p.shadow));
    checkCudaErrors(hipFree(p.state));
    checkCudaErrors(hipFree(p.attentuation));
    checkCudaErrors(hipFree(p.emitted));
    checkCudaErrors(hipFree(p.bounce));
    checkCudaErrors(hipFree(p.pstate));
    checkCudaErrors(hipFree(p.hit_id));
    checkCudaErrors(hipFree(p.hit_normal));
    checkCudaErrors(hipFree(p.hit_t));
    checkCudaErrors(hipFree(p.next_sample));

    checkCudaErrors(hipFree(p.active_paths));
    checkCudaErrors(hipFree(p.next_path));

    p.m.freeDeviceMem();
    checkCudaErrors(hipFree(p.numsamples_perpixel));
}

__global__ void fetch_samples(const render_params params, paths p, bool first, const camera cam) {
    // kMaxActivePaths threads are started to fetch the samples from all_sample_pool and initialize the paths
    // to keep things simple a block contains a single warp so that we only need to keep a single shared nextSample per block

    const unsigned int pid = threadIdx.x + blockIdx.x * blockDim.x;
    if (pid == 0)
        p.next_path[0] = 0;
    p.m.reset(pid, first);
    __syncthreads();

    if (pid >= params.maxActivePaths)
        return;

    rand_state state;
    pathstate pstate;
    if (first) {
        // this is the very first init, all paths are marked terminated, and we don't have a valid random state yet
        state = (wang_hash(pid) * 336343633) | 1;
        pstate = DONE;
    } else {
        state = p.state[pid];
        pstate = p.pstate[pid];
    }

    // generate all terminated paths
    const bool          terminated     = pstate == DONE;
    const unsigned int  maskTerminated = __ballot_sync(__activemask(), terminated);
    const int           numTerminated  = __popc(maskTerminated);
    const int           idxTerminated  = __popc(maskTerminated & ((1u << threadIdx.x) - 1));

    __shared__ volatile ull nextSample;

    if (terminated) {
        // first terminated lane increments next_sample
        if (idxTerminated == 0)
            nextSample = atomicAdd(p.next_sample, numTerminated);

        // compute sample this lane is going to fetch
        const ull sample_id = nextSample + idxTerminated;
        //const ull max_samples = (ull)params.width * (ull)params.height * (ull)params.spp;
        //if (sample_id >= max_samples)
        //    return; // no more samples to fetch

        // retrieve pixel_id corresponding to current path
        const unsigned int pixel_id = (sample_id / params.spp) % (params.width * params.height);
        p.active_paths[pid] = pixel_id;
        atomicAdd(p.numsamples_perpixel + pixel_id, 1);

        // compute pixel coordinates
        const unsigned int x = pixel_id % params.width;
        const unsigned int y = pixel_id / params.width;

        // generate camera ray
        float u = float(x + random_float(state)) / float(params.width);
        float v = float(y + random_float(state)) / float(params.height);
        p.r[pid] = cam.get_ray(u, v, state);
        p.state[pid] = state;
        p.attentuation[pid] = vec3(1, 1, 1);
        p.bounce[pid] = 0;
        p.pstate[pid] = SCATTER;
    }
}

#define IDX_SENTINEL    0
#define IS_DONE(idx)    (idx == IDX_SENTINEL)
#define IS_LEAF(idx)    (idx >= params.leaf_offset)

#define BIT_DONE        3
#define BIT_MASK        3
#define BIT_PARENT      0
#define BIT_LEFT        1
#define BIT_RIGHT       2

__device__ void pop_bitstack(unsigned long long& bitstack, int& idx) {
    const int m = (__ffsll(bitstack) - 1) / 2;
    bitstack >>= (m << 1);
    idx >>= m;

    if (bitstack == BIT_DONE) {
        idx = IDX_SENTINEL;
    }
    else {
        // idx could point to left or right child regardless of sibling we need to go to
        idx = (idx >> 1) << 1; // make sure idx always points to left sibling
        idx += (bitstack & BIT_MASK) - 1; // move idx to the sibling stored in bitstack
        bitstack = bitstack & (~BIT_MASK); // set bitstack to parent, so we can backtrack
    }
}

__global__ void trace_scattered(const render_params params, paths p) {
    // a limited number of threads are started to operate on active_paths

    unsigned int pid = 0; // currently traced path
    ray r; // corresponding ray

    // bvh traversal state
    int idx = IDX_SENTINEL;
    bool found;
    float closest;
    hit_record rec;

    unsigned long long bitstack;

    // Initialize persistent threads.
    // given that each block is 32 thread wide, we can use threadIdx.x as a warpId
    __shared__ volatile int nextPathArray[MaxBlockHeight]; // Current ray index in global buffer.
    __shared__ volatile bool noMorePaths[MaxBlockHeight]; // true when no more paths are available to fetch

    // Persistent threads: fetch and process rays in a loop.

    while (true) {
        const int tidx = threadIdx.x;
        volatile int& pathBase = nextPathArray[threadIdx.y];
        volatile bool& noMoreP = noMorePaths[threadIdx.y];
        pathstate pstate;

        // identify which lanes are done
        const bool          terminated      = IS_DONE(idx);
        const unsigned int  maskTerminated  = __ballot_sync(__activemask(), terminated);
        const int           numTerminated   = __popc(maskTerminated);
        const int           idxTerminated   = __popc(maskTerminated & ((1u << tidx) - 1));

        if (terminated) {
            // first terminated lane updates the base ray index
            if (idxTerminated == 0) {
                pathBase = atomicAdd(p.next_path, numTerminated);
                noMoreP = (pathBase + numTerminated) >= params.maxActivePaths;
            }

            pid = pathBase + idxTerminated;
            if (pid >= params.maxActivePaths) {
                return;
            }

            found = false; // always reset found to avoid writing hit information for terminated paths
            // setup ray if path not already terminated
            pstate = p.pstate[pid];
            if (pstate == SCATTER) {
                // Fetch ray
                r = p.r[pid];

                // idx is already set to IDX_SENTINEL, but make sure we set found to false
                idx = 1;
                closest = FLT_MAX;
                bitstack = BIT_DONE;
            }
        }

        // traversal
        while (!IS_DONE(idx)) {
            //p.m.lanes_cnt.increment(tidx);

            // we already intersected ray with idx node, now we need to load its children and intersect the ray with them
            if (!IS_LEAF(idx)) {
                // load left, right nodes
                bvh_node left, right;
                const int idx2 = idx * 2; // we are going to load and intersect children of idx
                if (idx2 < 2048) {
                    left = d_nodes[idx2];
                    right = d_nodes[idx2 + 1];
                }
                else {
                    // each spot in the texture holds two children, that's why we devide the relative texture index by 2
                    unsigned int tex_idx = ((idx2 - 2048) >> 1) * 3;
                    float4 a = tex1Dfetch(t_bvh, tex_idx++);
                    float4 b = tex1Dfetch(t_bvh, tex_idx++);
                    float4 c = tex1Dfetch(t_bvh, tex_idx++);
                    left = bvh_node(a.x, a.y, a.z, a.w, b.x, b.y);
                    right = bvh_node(b.z, b.w, c.x, c.y, c.z, c.w);
                }

                const float left_t = hit_bbox(left, r, closest);
                const bool traverse_left = left_t < FLT_MAX;
                const float right_t = hit_bbox(right, r, closest);
                const bool traverse_right = right_t < FLT_MAX;

                const bool swap = right_t < left_t; // right child is closer

                if (traverse_left || traverse_right) {
                    idx = idx2 + swap; // intersect closer node next
                    if (traverse_left && traverse_right) // push farther node into the stack
                        bitstack = (bitstack << 2) + (swap ? BIT_LEFT : BIT_RIGHT);
                    else // push parent bit to the stack to backtrack later
                        bitstack = (bitstack << 2) + BIT_PARENT;
                }
                else {
                    pop_bitstack(bitstack, idx);
                }
            } else {
                int m = (idx - params.leaf_offset) * lane_size_float;
                #pragma unroll
                for (int i = 0; i < lane_size_spheres; i++) {
                    float x = tex1Dfetch(t_spheres, m++);
                    float y = tex1Dfetch(t_spheres, m++);
                    float z = tex1Dfetch(t_spheres, m++);
                    vec3 center(x, y, z);
                    if (hit_point(center, r, 0.001f, closest, rec)) {
                        found = true;
                        closest = rec.t;
                        rec.idx = (idx - params.leaf_offset) * lane_size_spheres + i;
                    }
                }

                if (found) // exit traversal once we find an intersection in any leaf
                    idx = IDX_SENTINEL;
                else
                    pop_bitstack(bitstack, idx);
            }

            // some lanes may have already exited the loop, if not enough active thread are left, exit the loop
            if (!noMoreP && __popc(__activemask()) < DYNAMIC_FETCH_THRESHOLD)
                break;
        }

        if (pstate == SCATTER && IS_DONE(idx)) {
            if (found) {
                // finished traversing bvh
                p.hit_id[pid] = rec.idx;
                p.hit_normal[pid] = rec.n;
                p.hit_t[pid] = rec.t;
                p.pstate[pid] = HIT;
            } else {
                p.pstate[pid] = NO_HIT;
            }
        }
    }
}

// generate shadow rays for all non terminated rays with intersections
__global__ void generate_shadow_rays(const render_params params, paths p) {

    const vec3 light_center(5000, 0, 0);
    const float light_radius = params.lightRadius;
    const vec3 light_color = params.lightColor;

    // kMaxActivePaths threads update all p.num_active_paths
    const unsigned int pid = threadIdx.x + blockIdx.x * blockDim.x;
    if (pid == 0)
        p.next_path[0] = 0;
    __syncthreads();

    if (pid >= params.maxActivePaths)
        return;

    // if the path has no intersection, which includes terminated paths, do nothing
    if (p.pstate[pid] != HIT)
        return;

    const ray r = p.r[pid];
    const float hit_t = p.hit_t[pid];
    const vec3 hit_p = r.point_at_parameter(hit_t);
    const vec3 hit_n = p.hit_normal[pid];
    rand_state state = p.state[pid];

    // create a random direction towards the light
    // coord system for sampling
    const vec3 sw = unit_vector(light_center - hit_p);
    const vec3 su = unit_vector(cross(fabs(sw.x()) > 0.01f ? vec3(0, 1, 0) : vec3(1, 0, 0), sw));
    const vec3 sv = cross(sw, su);

    // sample sphere by solid angle
    const float cosAMax = sqrt(1.0f - light_radius * light_radius / (hit_p - light_center).squared_length());
    const float eps1 = random_float(state);
    const float eps2 = random_float(state);
    const float cosA = 1.0f - eps1 + eps1 * cosAMax;
    const float sinA = sqrt(1.0f - cosA * cosA);
    const float phi = 2 * kPI * eps2;
    const vec3 l = unit_vector(su * cosf(phi) * sinA + sv * sinf(phi) * sinA + sw * cosA);

    p.state[pid] = state;
    const float dotl = dot(l, hit_n);
    if (dotl <= 0)
        return;

    const float omega = 2 * kPI * (1.0f - cosAMax);
    p.shadow[pid] = ray(hit_p, l);
    p.emitted[pid] = light_color * dotl * omega / kPI;
    p.pstate[pid] = SHADOW;
}

// traces all paths that have FLAG_HAS_SHADOW set, sets FLAG_SHADOW_HIT to true if there is a hit
__global__ void trace_shadows(const render_params params, paths p) {
    // a limited number of threads are started to operate on active_paths

    unsigned int pid = 0; // currently traced path
    ray r; // corresponding ray

    // bvh traversal state
    int idx = IDX_SENTINEL;
    bool found = false;
    hit_record rec;

    unsigned long long bitstack;

    // Initialize persistent threads.
    // given that each block is 32 thread wide, we can use threadIdx.x as a warpId
    __shared__ volatile int nextPathArray[MaxBlockHeight]; // Current ray index in global buffer.

    // Persistent threads: fetch and process rays in a loop.

    while (true) {
        const int tidx = threadIdx.x;
        volatile int& pathBase = nextPathArray[threadIdx.y];
        pathstate pstate;

        // identify which lanes are done
        const bool          terminated = IS_DONE(idx);
        const unsigned int  maskTerminated = __ballot_sync(__activemask(), terminated);
        const int           numTerminated = __popc(maskTerminated);
        const int           idxTerminated = __popc(maskTerminated & ((1u << tidx) - 1));

        if (terminated) {
            // first terminated lane updates the base ray index
            if (idxTerminated == 0)
                pathBase = atomicAdd(p.next_path, numTerminated);

            pid = pathBase + idxTerminated;
            if (pid >= params.maxActivePaths)
                return;

            // setup ray if path has a shadow ray
            pstate = p.pstate[pid];
            if (pstate == SHADOW) {
                // Fetch ray
                r = p.shadow[pid];

                // idx is already set to IDX_SENTINEL, but make sure we set found to false
                found = false;
                idx = 1;
                bitstack = BIT_DONE;
            }
        }

        // traversal
        while (!IS_DONE(idx)) {
            // we already intersected ray with idx node, now we need to load its children and intersect the ray with them
            if (!IS_LEAF(idx)) {
                // load left, right nodes
                bvh_node left, right;
                const int idx2 = idx * 2; // we are going to load and intersect children of idx
                if (idx2 < 2048) {
                    left = d_nodes[idx2];
                    right = d_nodes[idx2 + 1];
                }
                else {
                    // each spot in the texture holds two children, that's why we devide the relative texture index by 2
                    unsigned int tex_idx = ((idx2 - 2048) >> 1) * 3;
                    float4 a = tex1Dfetch(t_bvh, tex_idx++);
                    float4 b = tex1Dfetch(t_bvh, tex_idx++);
                    float4 c = tex1Dfetch(t_bvh, tex_idx++);
                    left = bvh_node(a.x, a.y, a.z, a.w, b.x, b.y);
                    right = bvh_node(b.z, b.w, c.x, c.y, c.z, c.w);
                }

                const float left_t = hit_bbox(left, r, FLT_MAX);
                const bool traverse_left = left_t < FLT_MAX;
                const float right_t = hit_bbox(right, r, FLT_MAX);
                const bool traverse_right = right_t < FLT_MAX;

                const bool swap = right_t < left_t; // right child is closer

                if (traverse_left || traverse_right) {
                    idx = idx2 + swap; // intersect closer node next
                    if (traverse_left && traverse_right) // push farther node into the stack
                        bitstack = (bitstack << 2) + (swap ? BIT_LEFT : BIT_RIGHT);
                    else // push parent bit to the stack to backtrack later
                        bitstack = (bitstack << 2) + BIT_PARENT;
                }
                else {
                    pop_bitstack(bitstack, idx);
                }
            }
            else {
                int m = (idx - params.leaf_offset) * lane_size_float;
                #pragma unroll
                for (int i = 0; i < lane_size_spheres && !found; i++) {
                    float x = tex1Dfetch(t_spheres, m++);
                    float y = tex1Dfetch(t_spheres, m++);
                    float z = tex1Dfetch(t_spheres, m++);
                    vec3 center(x, y, z);
                    found = hit_point(center, r, 0.001f, FLT_MAX, rec);
                }

                if (found) // exit traversal once we find an intersection in any leaf
                    idx = IDX_SENTINEL;
                else
                    pop_bitstack(bitstack, idx);
            }

            // some lanes may have already exited the loop, if not enough active thread are left, exit the loop
            if (__popc(__activemask()) < DYNAMIC_FETCH_THRESHOLD) {
                break;
            }
        }

        if (pstate == SHADOW)
            p.pstate[pid] = found ? HIT : HIT_AND_LIGHT;
    }
}

// http://chilliant.blogspot.com.au/2012/08/srgb-approximations-for-hlsl.html
__host__ __device__ uint32_t LinearToSRGB(float x)
{
    x = max(x, 0.0f);
    x = max(1.055f * powf(x, 0.416666667f) - 0.055f, 0.0f);
    uint32_t u = min((uint32_t)(x * 255.9f), 255u);
    return u;
}

// convert floating point rgb color to 8-bit integer
__device__ int rgbToInt(float r, float g, float b)
{
    r = LinearToSRGB(r);
    g = LinearToSRGB(g);
    b = LinearToSRGB(b);
    return (int(b) << 16) | (int(g) << 8) | int(r);
}

__global__ void copyToUintBuffer(const render_params params, ull* numsamples_perpixel, unsigned int* uint_render_buffer) {
    const unsigned int pixel_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (pixel_id >= (params.width * params.height))
        return;
    const vec3 pixel = params.fb[pixel_id];
    const ull spp = numsamples_perpixel[pixel_id];
    uint_render_buffer[pixel_id] = rgbToInt(pixel.r() / spp, pixel.g() / spp, pixel.b() / spp);
}

// for all non terminated rays, accounts for shadow hit, compute scattered ray and resets the flag
__global__ void update(const render_params params, paths p) {
    const vec3 sky_emissive = params.skyColor;

    // kMaxActivePaths threads update all p.num_active_paths
    const unsigned int pid = threadIdx.x + blockIdx.x * blockDim.x;
    if (pid >= params.maxActivePaths)
        return;

    // is the path already done ?
    pathstate pstate = p.pstate[pid];
    if (pstate == DONE)
        return; // yup, done and already taken care of
    unsigned short bounce = p.bounce[pid];

    // did the ray hit a primitive ?
    if (pstate == HIT || pstate == HIT_AND_LIGHT) {
        // update path attenuation
        const int hit_id = p.hit_id[pid];
        int clr_idx = params.colors[hit_id] * 3;
        const vec3 albedo = vec3(d_colormap[clr_idx++], d_colormap[clr_idx++], d_colormap[clr_idx++]);
        
        vec3 attenuation = p.attentuation[pid] * albedo;
        p.attentuation[pid] = attenuation;

        // account for light contribution if no shadow hit
        if (pstate == HIT_AND_LIGHT) {
            const vec3 incoming = p.emitted[pid] * attenuation;
            const unsigned int pixel_id = p.active_paths[pid];
            atomicAdd(params.fb[pixel_id].e, incoming.e[0]);
            atomicAdd(params.fb[pixel_id].e + 1, incoming.e[1]);
            atomicAdd(params.fb[pixel_id].e + 2, incoming.e[2]);
        }

        // scatter ray, only if we didn't reach kMaxBounces
        bounce++;
        if (bounce < params.maxBounces) {
            const ray r = p.r[pid];
            const float hit_t = p.hit_t[pid];
            const vec3 hit_p = r.point_at_parameter(hit_t);

            const vec3 hit_n = p.hit_normal[pid];
            rand_state state = p.state[pid];
            const vec3 target = hit_n + random_in_unit_sphere(state);

            p.r[pid] = ray(hit_p, target);
            p.state[pid] = state;
            pstate = SCATTER;
        } else {
            pstate = DONE;
        }
    }
    else {
        if (bounce > 0) {
            const vec3 incoming = p.attentuation[pid] * sky_emissive;
            const unsigned int pixel_id = p.active_paths[pid];
            atomicAdd(params.fb[pixel_id].e, incoming.e[0]);
            atomicAdd(params.fb[pixel_id].e + 1, incoming.e[1]);
            atomicAdd(params.fb[pixel_id].e + 2, incoming.e[2]);
        }
        pstate = DONE;
    }

    p.pstate[pid] = pstate;
    p.bounce[pid] = bounce;
}

__global__ void print_metrics(metrics m, unsigned int iteration, unsigned int maxActivePaths, float elapsedSeconds, bool last) {
    m.print(iteration, elapsedSeconds, last);
}

void copySceneToDevice(const scene& sc, int** d_colors) {
    // copy the first 2048 nodes to constant memory
    const int const_size = min(2048, sc.bvh_size);
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_nodes), sc.bvh, const_size * sizeof(bvh_node)));

    // copy remaining nodes to global memory
    int remaining = sc.bvh_size - const_size;
    if (remaining > 0) {
        // declare and allocate memory
        const int buf_size_bytes = remaining * 6 * sizeof(float);
        checkCudaErrors(hipMalloc(&d_bvh_buf, buf_size_bytes));
        checkCudaErrors(hipMemcpy(d_bvh_buf, (void*)(sc.bvh + const_size), buf_size_bytes, hipMemcpyHostToDevice));
        checkCudaErrors(hipBindTexture(NULL, t_bvh, (void*)d_bvh_buf, buf_size_bytes));
    }

    // copying spheres to texture memory
    const int spheres_size_float = lane_size_float * (sc.spheres_size / lane_size_spheres);

    // copy the spheres in array of floats
    // do it after we build the BVH as it would have moved the spheres around
    float* floats = new float[spheres_size_float];
    int* colors = new int[sc.spheres_size];
    int idx = 0;
    int i = 0;
    while (i < sc.spheres_size) {
        for (int j = 0; j < lane_size_spheres; j++, i++) {
            floats[idx++] = sc.spheres[i].center.x();
            floats[idx++] = sc.spheres[i].center.y();
            floats[idx++] = sc.spheres[i].center.z();
            colors[i] = sc.spheres[i].color;
        }
        idx += lane_padding_float; // padding
    }
    assert(idx == scene_size_float);

    checkCudaErrors(hipMalloc((void**)d_colors, sc.spheres_size * sizeof(int)));
    checkCudaErrors(hipMemcpy(*d_colors, colors, sc.spheres_size * sizeof(int), hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void**)& d_spheres_buf, spheres_size_float * sizeof(float)));
    checkCudaErrors(hipMemcpy(d_spheres_buf, floats, spheres_size_float * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipBindTexture(NULL, t_spheres, (void*)d_spheres_buf, spheres_size_float * sizeof(float)));

    delete[] floats;
    delete[] colors;
}

void releaseScene(int* d_colors) {
    // destroy texture object
    checkCudaErrors(hipUnbindTexture(t_bvh));
    checkCudaErrors(hipUnbindTexture(t_spheres));
    checkCudaErrors(hipFree(d_bvh_buf));
    checkCudaErrors(hipFree(d_spheres_buf));
    checkCudaErrors(hipFree(d_colors));
}

void setup_camera(int nx, int ny, float dist) {
    vec3 lookfrom(dist, dist, dist);
    vec3 lookat(0, 0, 0);
    float dist_to_focus = (lookfrom - lookat).length();
    float aperture = 0.1;
    cam = new camera(lookfrom,
        lookat,
        vec3(0, 1, 0),
        30.0,
        float(nx) / float(ny),
        aperture,
        dist_to_focus);
    cam->look_from(c_theta, c_phi, c_relative_dist);
}

void write_image(const char* output_file, const vec3 *fb, const int nx, const int ny, const int ns) {
    unsigned int* idata = new unsigned int[nx * ny];
    checkCudaErrors(hipMemcpy(idata, d_cuda_render_buffer, nx * ny * sizeof(unsigned int), hipMemcpyDeviceToHost));

    char* data = new char[nx * ny * 3];
    int idx = 0;
    for (size_t i = 0; i < nx * ny; i++) {
        unsigned int pixel = idata[i];
        data[idx++] = pixel & 0xFF;
        data[idx++] = (pixel & 0xFF00) >> 8;
        data[idx++] = (pixel & 0xFF0000) >> 16;
    }
    stbi_write_png(output_file, nx, ny, 3, (void*)data, nx * 3);

    delete[] idata;
    delete[] data;
}

int cmpfunc(const void * a, const void * b) {
    if (*(double*)a > *(double*)b)
        return 1;
    else if (*(double*)a < *(double*)b)
        return -1;
    else
        return 0;
}

void initCuda(const options opt) {
    int num_pixels = opt.nx * opt.ny;
    size_t fb_size = num_pixels * sizeof(vec3);
    checkCudaErrors(hipMalloc((void**)&d_fb, fb_size));
    checkCudaErrors(hipMemset(d_fb, 0, fb_size));
}

void loadColormap(const char* filename) {
    vector<vector<float>> data = parse2DCsvFile(filename);
    float* colormap = new float[data.size() * 3];
    int idx = 0;
    for (auto l : data) {
        colormap[idx++] = (float)l[0];
        colormap[idx++] = (float)l[1];
        colormap[idx++] = (float)l[2];
    }

    // copy colors to constant memory
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_colormap), colormap, 256 * 3 * sizeof(float)));
    delete[] colormap;
}

void saveImage(const options opt, const char* filename) {
    int num_pixels = opt.nx * opt.ny;
    size_t fb_size = num_pixels * sizeof(vec3);
    vec3* h_fb = new vec3[fb_size];
    checkCudaErrors(hipMemcpy(h_fb, d_fb, fb_size, hipMemcpyDeviceToHost));
    write_image(filename, h_fb, opt.nx, opt.ny, opt.ns);
    delete[] h_fb;
}

int loadScene(const options opt) {
    scene sc;
    if (!opt.binary) {
        load_from_csv(opt.input, sc);
        store_to_binary(strcat(opt.input, ".bin"), sc);
    }
    else {
        load_from_binary(opt.input, sc);
    }
    copySceneToDevice(sc, &d_colors);
    sc.release();

    return sc.bvh_size;
}

render_params setupRenderParams(options opt, int bvh_size) {
    render_params params;
    params.fb = d_fb;
    params.leaf_offset = bvh_size / 2;
    params.colors = d_colors;
    params.width = opt.nx;
    params.height = opt.ny;
    params.spp = opt.ns;
    params.maxActivePaths = opt.maxActivePaths;
    return params;
}

void renderIteration(const options& opt, const render_params& params, const paths& p, const camera& cam, unsigned int iteration, bool lightEnabled) {

    // init kMaxActivePaths using equal number of threads
    {
        const int threads = 32; // 1 warp per block
        const int blocks = (opt.maxActivePaths + threads - 1) / threads;
        fetch_samples <<< blocks, threads >>> (params, p, iteration == 0, cam);
        checkCudaErrors(hipGetLastError());
    }

    // traverse bvh
    {
        dim3 blocks(6400 * 2, 1);
        dim3 threads(MaxBlockWidth, MaxBlockHeight);
        trace_scattered <<< blocks, threads >>> (params, p);
        checkCudaErrors(hipGetLastError());
    }

    // generate shadow rays
    if (lightEnabled)
    {
        const int threads = 128;
        const int blocks = (opt.maxActivePaths + threads - 1) / threads;
        generate_shadow_rays <<< blocks, threads >>> (params, p);
        checkCudaErrors(hipGetLastError());
    }

    // trace shadow rays
    if (lightEnabled)
    {
        dim3 blocks(6400 * 2, 1);
        dim3 threads(MaxBlockWidth, MaxBlockHeight);
        trace_shadows <<< blocks, threads >>> (params, p);
        checkCudaErrors(hipGetLastError());
    }

    // update paths accounting for intersection and light contribution
    {
        const int threads = 128;
        const int blocks = (opt.maxActivePaths + threads - 1) / threads;
        update <<< blocks, threads >>> (params, p);
        checkCudaErrors(hipGetLastError());
    }
}

void resetRenderer() {
    r_iteration = 0;
    checkCudaErrors(hipMemset(d_fb, 0, r_num_pixels * sizeof(vec3)));
    checkCudaErrors(hipMemset((void*)r_paths->next_sample, 0, sizeof(ull)));
    checkCudaErrors(hipMemset((void*)r_paths->numsamples_perpixel, 0, r_num_pixels * sizeof(ull)));
}

void render(const options& opt, render_params& params, const paths& p, camera& cam) {
    static GuiParams guiParams;
    bool guiChanged = true;
    bool lightEnabled = true;
    clock_t start = clock();
    hipProfilerStart();

    while (!pollWindowEvents()) {
        if (camera_updated || guiChanged) {
            cam.look_from(c_theta, c_phi, c_relative_dist);
            resetRenderer();
            camera_updated = false;
            guiChanged = false;

            params.maxBounces = guiParams.maxBounces;
            params.lightRadius = guiParams.lightRadius;
            params.lightColor = vec3(guiParams.lightColor[0], guiParams.lightColor[1], guiParams.lightColor[2]) * guiParams.lightIntensity;
            lightEnabled = guiParams.lightIntensity > 0;
            params.skyColor = vec3(guiParams.skyColor[0], guiParams.skyColor[1], guiParams.skyColor[2]) * guiParams.skyIntensity;
        }

        renderIteration(opt, params, p, cam, r_iteration, lightEnabled);

        {
            const int threads = 128;
            const int blocks = (params.width * params.height + threads - 1) / threads;
            copyToUintBuffer <<< blocks, threads >>> (params, p.numsamples_perpixel, d_cuda_render_buffer);

            updateWindow(guiParams, guiChanged);
        }

        // print metrics
        if (opt.verbose) {
            print_metrics <<< 1, 1 >>> (p.m, r_iteration, opt.maxActivePaths, (float)(clock() - start) / CLOCKS_PER_SEC, false);
            checkCudaErrors(hipGetLastError());
        }

        r_iteration++;
    }
    hipProfilerStop();

    print_metrics <<< 1, 1 >>> (p.m, r_iteration, opt.maxActivePaths, (float)(clock() - start) / CLOCKS_PER_SEC, true);
    checkCudaErrors(hipGetLastError());

    checkCudaErrors(hipDeviceSynchronize());
}

void mouseMove(int dx, int dy, int mouse_btn) {
    if (mouse_btn == MOUSE_LEFT) {
        c_theta += -dy * c_rotation_speed;
        //if (theta < delta) theta = delta;
        //if (theta > (kPI/2 - delta)) theta = kPI/2 - delta;
        c_phi += -dx * c_rotation_speed;
    }
    else {
        // drag with right button changes camera distance
        // only x movement is taken into account
        c_relative_dist += dx * c_zoom_speed;
        if (c_relative_dist < 0.1f)
            c_relative_dist = 0.1f;
    }
    camera_updated = true;
}

int main(int argc, char** argv) {
    options opt;
    if (!parse_args(argc, argv, opt))
        return -1;

    initWindow(argc, argv, opt.nx, opt.ny, &d_cuda_render_buffer);
    registerMouseMoveFunc(mouseMove);

    initCuda(opt);
    loadColormap(opt.colormap);

    r_num_pixels = opt.nx * opt.ny;

    const int bvh_size = loadScene(opt);

    setup_camera(opt.nx, opt.ny, opt.dist);

    render_params params = setupRenderParams(opt, bvh_size);

    r_paths = new paths();
    setup_paths(*r_paths, opt.nx, opt.ny, opt.ns, opt.maxActivePaths);

    cout << "started renderer\n" << std::flush;

    render(opt, params, *r_paths, *cam);

    char imagename[100];
    sprintf(imagename, "%s_%dx%dx%d_%d_bvh.png", opt.input, opt.nx, opt.ny, opt.ns, opt.dist);
    saveImage(opt, imagename);

    // clean up
    destroyWindow();

    delete cam;
    free_paths(*r_paths);
    delete r_paths;
    releaseScene(d_colors);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(params.fb));

    hipDeviceReset();
}