#include "hip/hip_runtime.h"
#include <float.h>

#include "ray.h"
#include "camera.h"
#include "scene.h"

#define STBI_MSC_SECURE_CRT
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

const int kMaxBounces = 10;
const int nx = 1200;
const int ny = 1200;

// Matching the C++ code would recurse enough into color() calls that
// it was blowing up the stack, so we have to turn this into a
// limited-depth loop instead.  Later code in the book limits to a max
// depth of 50, so we adapt this a few chapters early on the GPU.
__device__ vec3 color(const ray& r, const scene s, rand_state& rand_state) {
    vec3 light_center(5000, 0, 0);
    float light_radius = 500;
    float light_emissive = 100;
    float sky_emissive = .2f;

    ray cur_ray = r;
    vec3 attenuation = vec3(1, 1, 1);
    vec3 incoming = vec3(0, 0, 0);
    for (int i = 0; i < kMaxBounces; i++) {
        hit_record rec;
        if (hit_bvh(s, cur_ray, 0.001f, FLT_MAX, rec)) {
            const vec3 p = cur_ray.point_at_parameter(rec.t);
            vec3 target = rec.n + random_in_unit_sphere(rand_state);

            int clr_idx = s.colors[rec.idx] * 3;
            vec3 albedo = vec3(d_colormap[clr_idx++], d_colormap[clr_idx++], d_colormap[clr_idx++]);

            // explicit light sampling

            // create a random direction towards sphere
            
            // coord system for sampling: sw, su, sv
            vec3 sw = unit_vector(light_center - p);
            vec3 su = unit_vector(cross(fabs(sw.x()) > 0.01f ? vec3(0, 1, 0) : vec3(1, 0, 0), sw));
            vec3 sv = cross(sw, su);
            
            // sample sphere by solid angle
            float cosAMax = sqrtf(1.0f - light_radius * light_radius / (p - light_center).squared_length());
            float eps1 = random_float(rand_state), eps2 = random_float(rand_state);
            float cosA = 1.0f - eps1 + eps1 * cosAMax;
            float sinA = sqrtf(1.0f - cosA * cosA);
            float phi = 2 * kPI * eps2;
            vec3 l = unit_vector(su * cosf(phi) * sinA + sv * sin(phi) * sinA + sw * cosA);

            // shoot shadow ray
            if (!shadow_bvh(s, ray(p, l), 0.001f, FLT_MAX)) {
                float omega = 2 * kPI * (1 - cosAMax);

                vec3 rdir = cur_ray.direction();
                vec3 nl = dot(rec.n, rdir) < 0 ? rec.n : -rec.n;
                incoming += attenuation * (albedo * light_emissive) * (fmaxf(0.0f, dot(l, nl)) * omega / kPI);
            }

            attenuation *= albedo;
            cur_ray = ray(p, target);
        }
        else if (i == 0) { // primary ray didn't hit anything
            break; // black background
        }
        else {
            return incoming + attenuation * sky_emissive;
        }
    }
    return incoming; // exceeded recursion
}

__global__ void render(vec3 *fb, const scene sc, int max_x, int max_y, int ns, int frame, const camera cam) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    rand_state state = ((wang_hash(pixel_index) + frame * 101141101) * 336343633) | 1;
    vec3 col(0, 0, 0);
    for (int s = 0; s < ns; s++) {
        float u = float(i + random_float(state)) / float(max_x);
        float v = float(j + random_float(state)) / float(max_y);
        ray r = cam.get_ray(u, v, state);
        col += color(r, sc, state);
    }
    fb[pixel_index] += col;
}

float rand(unsigned int &state) {
    state = (214013 * state + 2531011);
    return (float)((state >> 16) & 0x7FFF) / 32767;
}

#define RND (rand(rand_state))


camera setup_camera(int nx, int ny, float dist) {
    vec3 lookfrom(dist, dist, dist);
    vec3 lookat(0, 0, 0);
    float dist_to_focus = (lookfrom - lookat).length();
    float aperture = 0.1;
    return camera(lookfrom,
        lookat,
        vec3(0, 1, 0),
        30.0,
        float(nx) / float(ny),
        aperture,
        dist_to_focus);
}

// http://chilliant.blogspot.com.au/2012/08/srgb-approximations-for-hlsl.html
static uint32_t LinearToSRGB(float x)
{
    x = max(x, 0.0f);
    x = max(1.055f * powf(x, 0.416666667f) - 0.055f, 0.0f);
    uint32_t u = min((uint32_t)(x * 255.9f), 255u);
    return u;
}

void write_image(const char* output_file, const vec3 *fb, const int nx, const int ny, const int ns) {
    char *data = new char[nx * ny * 3];
    int idx = 0;
    for (int j = ny - 1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j * nx + i;
            data[idx++] = LinearToSRGB(fb[pixel_index].r() / ns);
            data[idx++] = LinearToSRGB(fb[pixel_index].g() / ns);
            data[idx++] = LinearToSRGB(fb[pixel_index].b() / ns);
        }
    }
    stbi_write_png(output_file, nx, ny, 3, (void*)data, nx * 3);
    delete[] data;
}

int cmpfunc(const void * a, const void * b) {
    if (*(double*)a > *(double*)b)
        return 1;
    else if (*(double*)a < *(double*)b)
        return -1;
    else
        return 0;
}

int main(int argc, char** argv) {
    if (argc < 2) {
        cerr << "usage spheres file_name [num_samples=1] [num_runs=1] [camera_dist=100] [colormap=viridis.csv]";
        exit(-1);
    }
    char* input = argv[1];
    const int ns = (argc > 2) ? strtol(argv[2], NULL, 10) : 1;
    const int tx = 8;
    const int ty = 8;
    int nr = (argc > 3) ? strtol(argv[3], NULL, 10) : 1;
    if (nr == 0) nr = INT_MAX;
    const int dist = (argc > 4) ? strtof(argv[4], NULL) : 100;
    const char* colormap = (argc > 5) ? argv[5] : "viridis.csv";

    const bool is_csv = strncmp(input + strlen(input) - 4, ".csv", 4) == 0;
    
    cerr << "Rendering a " << nx << "x" << ny << " image with " << ns << " samples per pixel ";
    cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx * ny;
    size_t fb_size = num_pixels * sizeof(vec3);

    // allocate FB
    vec3 *d_fb;
    checkCudaErrors(hipMalloc((void **)&d_fb, fb_size));
    checkCudaErrors(hipMemset(d_fb, 0, fb_size));

    // load colormap
    vector<vector<float>> data = parse2DCsvFile(colormap);
    cout << "colormap contains " << data.size() << " points\n";
    float *_viridis_data = new float[data.size() * 3];
    int idx = 0;
    for (auto l : data) {
        _viridis_data[idx++] = (float)l[0];
        _viridis_data[idx++] = (float)l[1];
        _viridis_data[idx++] = (float)l[2];
    }
    // setup scene
    scene sc;
    setup_scene(input, sc, is_csv, _viridis_data);
    delete[] _viridis_data;
    _viridis_data = NULL;

    camera cam = setup_camera(nx, ny, dist);
    vec3* h_fb = new vec3[fb_size];

    double render_time = 0;
    for (int r = 0, frame = 0; r < nr; r++, frame += ns) {
        // Render our buffer
        clock_t start;
        start = clock();
        dim3 blocks(nx / tx + 1, ny / ty + 1);
        dim3 threads(tx, ty);
        render << <blocks, threads >> >(d_fb, sc, nx, ny, ns, frame, cam);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());
        render_time += clock() - start;
        cerr << "rendered " << (frame + ns) << " samples in " << render_time / CLOCKS_PER_SEC << " seconds.\r";

        // save temp output
        checkCudaErrors(hipMemcpy(h_fb, d_fb, fb_size, hipMemcpyDeviceToHost));
        write_image("inprogress.png", h_fb, nx, ny, frame + ns);
    }

    // Output FB as Image
    checkCudaErrors(hipMemcpy(h_fb, d_fb, fb_size, hipMemcpyDeviceToHost));
    char file_name[100];
    sprintf(file_name, "%s_%dx%dx%d_%d_bvh.png", input, nx, ny, ns*nr, dist);
    write_image(file_name, h_fb, nx, ny, ns*nr);
    delete[] h_fb;
    h_fb = NULL;

    // clean up
    releaseScene(sc);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_fb));

    hipDeviceReset();
}